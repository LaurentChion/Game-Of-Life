#include "hip/hip_runtime.h"
#include "Grid.hpp"

namespace Step {

  // fonction qui fonctionne mais on n'utilise pas la classe grille en parametre
  __global__ void cudaStep(int * gridIn, int * gridOut, int dimRow, int dimColumn) {
    int threadId =  blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < dimRow * dimColumn) {
      int i = threadId / dimColumn;
      int j = threadId % dimColumn;

      int top=0, bottom=0, left=0, right=0, count=0;

      top = (i-1 != -1) ? i-1 : dimRow - 1;
      bottom = (i+1 != dimRow) ? i+1 : 0;
      left = (j-1 != -1) ? j-1 : dimColumn - 1;
      right = (j+1 != dimColumn) ? j+1 : 0;

      count += (gridIn[top*dimColumn + left] == 1) ? 1 : 0;
      count += (gridIn[top*dimColumn + j] == 1) ? 1 : 0;
      count += (gridIn[top*dimColumn + right] == 1) ? 1 : 0;
      count += (gridIn[i*dimColumn + left] == 1) ? 1 : 0;

      count += (gridIn[i*dimColumn + right] == 1) ? 1 : 0;
      count += (gridIn[bottom*dimColumn + left] == 1) ? 1 : 0;
      count += (gridIn[bottom*dimColumn + j] == 1) ? 1 : 0;
      count += (gridIn[bottom*dimColumn + right] == 1) ? 1 : 0;

      if(gridIn[i*dimColumn + j] == 1) {
        if ( !(count == 2 || count == 3) ) {
          gridOut[i*dimColumn + j] = 0;
        }
      }
      else {
        if( count == 3) {
          gridOut[i*dimColumn + j] = 1;
        }
      }
    }
  }

  __global__ void cudaStep(Grid * gridIn, Grid * gridOut) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < gridIn->getDimRow() * gridIn->getDimColumn()) {
      int i = threadId / gridIn->getDimColumn();
      int j = threadId % gridIn->getDimColumn();

      int top=0, bottom=0, left=0, right=0, count=0;

      top = (i-1 != -1) ? i-1 : gridIn->getDimRow() - 1;
      bottom = (i+1 != gridIn->getDimRow()) ? i+1 : 0;
      left = (j-1 != -1) ? j-1 : gridIn->getDimColumn() - 1;
      right = (j+1 != gridIn->getDimColumn()) ? j+1 : 0;

      count += (gridIn->getTab(top, left) == 1) ? 1 : 0;
      count += (gridIn->getTab(top, j) == 1) ? 1 : 0;
      count += (gridIn->getTab(top, right) == 1) ? 1 : 0;
      count += (gridIn->getTab(i, left) == 1) ? 1 : 0;

      count += (gridIn->getTab(i, right) == 1) ? 1 : 0;
      count += (gridIn->getTab(bottom, left) == 1) ? 1 : 0;
      count += (gridIn->getTab(bottom, j) == 1) ? 1 : 0;
      count += (gridIn->getTab(bottom, right) == 1) ? 1 : 0;

      if(gridIn->getTab(i, j) == 1) {
        if ( !(count == 2 || count == 3) ) {
          gridOut->setTab(i, j, 0);
        }
      }
      else {
        if( count == 3) {
          gridOut->setTab(i, j, 1);
        }
      }
    }
  }

}
